#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../cellmodels/Land_2016.hpp"
#include "../cellmodels/Ohara_Rudy_2011.hpp"
#include "../utils/constants.hpp"
#include "glob_funct.hpp"
#include "glob_type.hpp"
#include "gpu.cuh"
#include "param.hpp"

/**
 * @brief Main kernel function to run drug simulation for all samples in parallel.
 *
 * @param d_ic50 Array of IC50 values.
 * @param d_cvar Array of conductance variability values.
 * @param d_conc Array of drug concentrations.
 * @param d_CONSTANTS Array of constants.
 * @param d_STATES Array of states.
 * @param d_RATES Array of rates.
 * @param d_ALGEBRAIC Array of algebraic values.
 * @param d_STATES_RESULT Array to store the result states.
 * @param sample_size Sample size.
 * @param temp_result Temporary result array.
 * @param cipa_result CIPA result array.
 * @param p_param Parameters.
 */
__global__ void kernel_DrugSimulation(double *d_ic50, double *d_cvar, double *d_conc, double *d_CONSTANTS,
                                      double *d_STATES, double *d_STATES_init, double *d_RATES, double *d_ALGEBRAIC,
                                      double *d_mec_CONSTANTS, double *d_mec_RATES, double *d_mec_STATES,
                                      double *d_mec_ALGEBRAIC, double *d_STATES_RESULT, double *time, double *states,
                                      double *out_dt, double *cai_result, double *ina, double *inal, double *ical,
                                      double *ito, double *ikr, double *iks, double *ik1, unsigned int sample_size,
                                      cipa_t *temp_result, cipa_t *cipa_result, param_t *p_param) {
    unsigned short thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id >= sample_size) return;

    // Local arrays for each sample
    double time_for_each_sample[10000];
    double dt_for_each_sample[10000];

    // Run the drug simulation for each sample
    kernel_DoDrugSim_init(d_ic50, d_cvar, d_conc[thread_id], d_CONSTANTS, d_STATES, d_RATES, d_ALGEBRAIC,
                          d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, d_STATES_RESULT,
                          time_for_each_sample, dt_for_each_sample, thread_id, sample_size, temp_result, cipa_result,
                          p_param);
}

/**
 * @brief Runs a single drug simulation on the GPU for a given sample.
 *
 * @param d_ic50 Array of IC50 values.
 * @param d_cvar Array of conductance variability values.
 * @param d_conc Drug concentration.
 * @param d_CONSTANTS Array of constants.
 * @param d_STATES Array of states.
 * @param d_RATES Array of rates.
 * @param d_ALGEBRAIC Array of algebraic values.
 * @param d_STATES_RESULT Array to store the result states.
 * @param tcurr Current time array.
 * @param dt Time step array.
 * @param sample_id Sample ID.
 * @param sample_size Sample size.
 * @param temp_result Temporary result array.
 * @param cipa_result CIPA result array.
 * @param p_param Parameters.
 */
__device__ void kernel_DoDrugSim_init(double *d_ic50, double *d_cvar, double d_conc, double *d_CONSTANTS,
                                      double *d_STATES, double *d_RATES, double *d_ALGEBRAIC, double *d_STATES_RESULT,
                                      double *d_mec_CONSTANTS, double *d_mec_RATES, double *d_mec_STATES,
                                      double *d_mec_ALGEBRAIC, double *tcurr, double *dt, unsigned short sample_id,
                                      unsigned int sample_size, cipa_t *temp_result, cipa_t *cipa_result,
                                      param_t *p_param) {
    unsigned int input_counter = 0;

    // Initialize temporary result and CiPA result structures
    auto init_result = [](cipa_t &result, const double *STATES, unsigned int sample_id) {
        result.qnet = 0.;
        result.inal_auc = 0.;
        result.ical_auc = 0.;
        result.dvmdt_repol = -999;
        result.dvmdt_max = -999;
        result.vm_peak = -999;
        result.vm_valley = STATES[(sample_id * ORd_num_of_states) + V];
        result.vm_dia = -999;
        result.apd90 = 0.;
        result.apd50 = 0.;
        result.ca_peak = -999;
        result.ca_valley = STATES[(sample_id * ORd_num_of_states) + cai];
        result.ca_dia = -999;
        result.cad90 = 0.;
        result.cad50 = 0.;
    };

    // Initialize results for this sample
    init_result(temp_result[sample_id], d_STATES, sample_id);
    init_result(cipa_result[sample_id], d_STATES, sample_id);

    // Simulation variables
    bool is_peak = false;
    tcurr[sample_id] = 0.000001;
    dt[sample_id] = p_param->dt;
    double max_time_step = 0.1, time_point = 25.0;
    double dt_set;
    int cipa_datapoint = 0;
    unsigned short pace_count = 0;
    double t_peak_capture = 0.0;
    unsigned short pace_steepest = 0;
    bool init_states_captured = false;
    bool is_eligible_AP;
    const double bcl = p_param->bcl;
    const unsigned short pace_max = p_param->pace_max;
    const unsigned short last_drug_check_pace = p_param->find_steepest_start;
    double tmax = pace_max * bcl;
    double conc = d_conc;
    double type = p_param->celltype;
    double y[7] = {0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0};
    double epsilon = 10E-14;
    double vm_repol30, vm_repol90;

    // Initialize constants and apply drug effects
    initConsts(d_CONSTANTS, d_STATES, type, conc, d_ic50, d_cvar, p_param->is_dutta, p_param->is_cvar, bcl, sample_id);
    applyDrugEffect(d_CONSTANTS, conc, d_ic50, epsilon, sample_id);
    land_initConsts(false, false, y, d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, sample_id);

    d_CONSTANTS[BCL + (sample_id * ORd_num_of_constants)] = bcl;

    // Main simulation loop
    // dt_set = 0.001;
    while (tcurr[sample_id] < tmax) {
        // Compute rates
        coupledComputeRates(tcurr[sample_id], d_CONSTANTS, d_RATES, d_STATES, d_ALGEBRAIC, sample_id,
                     d_mec_RATES[TRPN + (sample_id * Land_num_of_rates)]);
        land_computeRates(tcurr[sample_id], d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, d_mec_ALGEBRAIC, y, sample_id);
        // Set time step (adaptive dt)
        //NOTE: Disabled in Margara
        dt_set = set_time_step(tcurr[sample_id], time_point, max_time_step, d_CONSTANTS, d_RATES, d_STATES, d_ALGEBRAIC,
                              sample_id);
        // dt_set = 0.005;
        // Check if within the same cycle
        if (floor((tcurr[sample_id] + dt_set) / bcl) == floor(tcurr[sample_id] / bcl)) {
            dt[sample_id] = dt_set;
        } else {
            // Handle end of pacing cycle
            dt[sample_id] = (floor(tcurr[sample_id] / bcl) + 1) * bcl - tcurr[sample_id];

            // Update temporary results if this is the steepest pace
            if (temp_result[sample_id].dvmdt_repol > cipa_result[sample_id].dvmdt_repol) {
                pace_steepest = pace_count;
                cipa_result[sample_id] = temp_result[sample_id];
                cipa_result[sample_id].ca_valley = d_STATES[(sample_id * ORd_num_of_states) + cai];
                cipa_result[sample_id].vm_valley = d_STATES[(sample_id * ORd_num_of_states) + V];
                is_peak = true;
                init_states_captured = false;
            } else {
                is_peak = false;
            }

            // Reset variables for next pacing cycle
            t_peak_capture = 0.0;
            init_result(temp_result[sample_id], d_STATES, sample_id);
            pace_count++;
            input_counter = 0;
            cipa_datapoint = 0;
            is_eligible_AP = false;

            // Debug output
            if (sample_id == 0) {
                printf("core: %d pace count: %d t: %lf, steepest: %d, dvmdt_repol: %lf, conc: %lf\n", sample_id,
                       pace_count, tcurr[sample_id], pace_steepest, cipa_result[sample_id].dvmdt_repol, conc);
            }
        }

        // Solve ODEs analytically
        solveAnalytical(d_CONSTANTS, d_STATES, d_ALGEBRAIC, d_RATES, dt[sample_id], sample_id);
        land_solveEuler(dt[sample_id], tcurr[sample_id], d_STATES[cai + (sample_id * ORd_num_of_states)] * 1000.,
                        d_mec_CONSTANTS, d_mec_RATES, d_mec_STATES, sample_id);

        // Perform checks in the last few pacing cycles
        if (pace_count >= pace_max - last_drug_check_pace) {
            if (tcurr[sample_id] > ((d_CONSTANTS[(sample_id * ORd_num_of_constants) + BCL] * pace_count) +
                                    (d_CONSTANTS[(sample_id * ORd_num_of_constants) + stim_start] + 2)) &&
                tcurr[sample_id] < ((d_CONSTANTS[(sample_id * ORd_num_of_constants) + BCL] * pace_count) +
                                    (d_CONSTANTS[(sample_id * ORd_num_of_constants) + stim_start] + 10)) &&
                abs(d_ALGEBRAIC[(sample_id * ORd_num_of_algebraic) + INa]) < 1) {
                if (d_STATES[(sample_id * ORd_num_of_states) + V] > temp_result[sample_id].vm_peak) {
                    temp_result[sample_id].vm_peak = d_STATES[(sample_id * ORd_num_of_states) + V];
                    if (temp_result[sample_id].vm_peak > 0) {
                        vm_repol30 = temp_result[sample_id].vm_peak -
                                     (0.3 * (temp_result[sample_id].vm_peak - temp_result[sample_id].vm_valley));
                        vm_repol90 = temp_result[sample_id].vm_peak -
                                     (0.9 * (temp_result[sample_id].vm_peak - temp_result[sample_id].vm_valley));
                        is_eligible_AP = true;
                        t_peak_capture = tcurr[sample_id];
                    } else {
                        is_eligible_AP = false;
                    }
                }
            } else if (tcurr[sample_id] > ((d_CONSTANTS[(sample_id * ORd_num_of_constants) + BCL] * pace_count) +
                                           (d_CONSTANTS[(sample_id * ORd_num_of_constants) + stim_start] + 10)) &&
                       is_eligible_AP) {
                if (d_RATES[(sample_id * ORd_num_of_rates) + V] > temp_result[sample_id].dvmdt_repol &&
                    d_STATES[(sample_id * ORd_num_of_states) + V] <= vm_repol30 &&
                    d_STATES[(sample_id * ORd_num_of_states) + V] >= vm_repol90) {
                    temp_result[sample_id].dvmdt_repol = d_RATES[(sample_id * ORd_num_of_rates) + V];
                }
            }

            // Capture initial states and data points if in the last few paces
            if ((pace_count >= pace_max - last_drug_check_pace) && (is_peak == true) && (pace_count < pace_max)) {
                if (!init_states_captured) {
                    for (int counter = 0; counter < ORd_num_of_states; counter++) {
                        d_STATES_RESULT[(sample_id * ORd_num_of_states) + counter] =
                            d_STATES[(sample_id * ORd_num_of_states) + counter];
                    }
                    init_states_captured = true;
                }

                input_counter += sample_size;
                cipa_datapoint++;
            }
        }
        tcurr[sample_id] += dt[sample_id];
    }
}
