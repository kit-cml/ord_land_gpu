/**
 * @file main.cu
 * @brief Main entry point for the Drug Simulation using CUDA
 */

#include <sys/stat.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "modules/glob_funct.hpp"
#include "modules/glob_type.hpp"
#include "modules/gpu.cuh"
#include "utils/constants.hpp"
#include "utils/file_operations.hpp"
#include "utils/gpu_operations.hpp"
#include "utils/timing.hpp"

/**
 * @brief Main function for running the drug simulation
 *
 * @param argc Number of command-line arguments
 * @param argv Array of command-line arguments
 * @return int Exit status of the program
 */
int main(int argc, char **argv) {
    param_t *p_param = new param_t();  // input data for CPU
    param_t *d_p_param;                // input data for GPU parsing
    p_param->init();
    edison_assign_params(argc, argv, p_param);

    double *ic50 = (double *)malloc(14 * sample_limit * sizeof(double));
    double *conc = (double *)malloc(sample_limit * sizeof(double));
    double *cvar = (double *)malloc(18 * sample_limit * sizeof(double));
    char *drug_name = get_drug_name(p_param->hill_file);
    double *d_ic50, *d_conc, *d_cvar, *d_ALGEBRAIC, *d_CONSTANTS, *d_RATES, *d_STATES, *d_STATES_RESULT, *d_STATES_init;
    double *d_mec_ALGEBRAIC, *d_mec_CONSTANTS, *d_mec_RATES, *d_mec_STATES;
    double *time, *dt, *states, *ical, *inal, *cai_result, *ina, *ito, *ikr, *iks, *ik1;
    cipa_t *temp_result, *cipa_result;

    int sample_size = get_IC50_data_from_file(p_param->hill_file, ic50, conc);
    int blocksPerGrid = (sample_size + threadsPerBlock - 1) / threadsPerBlock;
    printf("Sample size: %d\nSet GPU Number: %d\n", sample_size, p_param->gpu_index);

    hipSetDevice(p_param->gpu_index);

    if (p_param->is_cvar) {
        int cvar_sample = get_cvar_data_from_file(p_param->cvar_file, sample_size, cvar);
        printf("Reading: %d Conductance Variability samples\n", cvar_sample);
    }

    prepingGPUMemory(sample_size, d_ALGEBRAIC, d_CONSTANTS, d_RATES, d_STATES, d_mec_ALGEBRAIC, d_mec_CONSTANTS,
                     d_mec_RATES, d_mec_STATES, d_p_param, temp_result, cipa_result, d_STATES_RESULT, d_ic50, ic50,
                     d_conc, conc, p_param);

    tic();

    if (p_param->is_time_series == 1 ) {

        std::regex pattern("/([a-zA-Z0-9_\.]+)\.csv");
        std::smatch match;
        std::string fname = p_param->hill_file;
        std::regex_search(fname, match, pattern);
        
        printf("%s\n", match[1].str().c_str());

        printf("Using cached initial state from previous result!!!! \n\n");

        const unsigned int datapoint_size = p_param->sampling_limit;
        double *cache;
        cache = (double *)malloc((num_of_states + 2) * sample_limit * sizeof(double));

        double *d_ic50;
        double *d_conc;
        double *d_cvar;
        double *d_ALGEBRAIC;
        double *d_CONSTANTS;
        double *d_RATES;
        double *d_STATES;
        double *d_STATES_cache;
        double *d_mec_CONSTANTS, *d_mec_STATES, *d_mec_RATES, *d_mec_ALGEBRAIC;
        // actually not used but for now, this is only for satisfiying the GPU regulator parameters
        double *d_STATES_RESULT;
        double *d_all_states;

        double *time;
        double *dt;
        double *states;
        double *ical;
        double *inal;
        double *cai_result;
        double *ina;
        double *ito;
        double *ikr;
        double *iks;
        double *ik1;
        double *tension;
        cipa_t *temp_result, *cipa_result;

        static const int CALCIUM_SCALING = 1000000;
        static const int CURRENT_SCALING = 1000;

        int num_of_constants = 146;
        int num_of_states = 42;
        int num_of_algebraic = 199;
        int num_of_rates = 42;

        // snprintf(buffer, sizeof(buffer),
        //   "./drugs/bepridil/IC50_samples.csv"
        //   // "./drugs/bepridil/IC50_optimal.csv"
        //   // "./IC50_samples.csv"
        //   );

        int sample_size = get_IC50_data_from_file(p_param->hill_file, ic50, conc, drug_name);
        if (sample_size == 0)
            printf("Something problem with the IC50 file!\n");
        // else if(sample_size > 2000)
        //     printf("Too much input! Maximum sample data is 2000!\n");
        printf("Sample size: %d\n", sample_size);
        printf("Set GPU Number: %d\n", p_param->gpu_index);

        hipSetDevice(p_param->gpu_index);

        if (p_param->is_cvar == true) {
            int cvar_sample = get_cvar_data_from_file(p_param->cvar_file, sample_size, cvar);
            printf("Reading: %d Conductance Variability samples\n", cvar_sample);
        }

        printf("preparing GPU memory space \n");

        // char buffer_cvar[255];
        // snprintf(buffer_cvar, sizeof(buffer_cvar),
        // "./result/66_00.csv"
        // // "./drugs/optimized_pop_10k.csv"
        // );
        int cache_num = get_init_data_from_file(p_param->cache_file, cache);
        printf("Found cache for %d samples\n", cache_num);
        // note to self:
        // num of states+2 gave you at the very end of the file (pace number)
        // the very beginning -> the core number
        //   for (int z = 0; z <  num_of_states; z++) {printf("%lf\n", cache[z+1]);}
        //   printf("\n");
        //   for (int z = 0; z <  num_of_states; z++) {printf("%lf\n", cache[ 1*(num_of_states+2) + (z+2)]);}
        //   printf("\n");
        //   for (int z = 0; z <  num_of_states; z++) {printf("%lf\n", cache[ 2*(num_of_states+2) + (z+3)]);}
        // return 0 ;

        hipMalloc(&d_ALGEBRAIC, num_of_algebraic * sample_size * sizeof(double));
        hipMalloc(&d_CONSTANTS, num_of_constants * sample_size * sizeof(double));
        hipMalloc(&d_RATES, num_of_rates * sample_size * sizeof(double));
        hipMalloc(&d_STATES, num_of_states * sample_size * sizeof(double));
        hipMalloc(&d_STATES_cache, (num_of_states + 2) * sample_size * sizeof(double));
        hipMalloc(&d_mec_ALGEBRAIC, 24 * sample_size * sizeof(double));
        hipMalloc(&d_mec_CONSTANTS, 29 * sample_size * sizeof(double));
        hipMalloc(&d_mec_RATES, 7 * sample_size * sizeof(double));
        hipMalloc(&d_mec_STATES, 7 * sample_size * sizeof(double));

        hipMalloc(&d_p_param, sizeof(param_t));

        // prep for 1 cycle plus a bit (7000 * sample_size)
        hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
        hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));

        hipMalloc(&time, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&dt, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&states, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ical, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&inal, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&cai_result, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ina, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ito, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ikr, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&iks, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ik1, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&tension, sample_size * datapoint_size * sizeof(double));
        // hipMalloc(&d_STATES_RESULT, (num_of_states+1) * sample_size * sizeof(double));
        // hipMalloc(&d_all_states, num_of_states * sample_size * p_param->find_steepest_start * sizeof(double));

        printf("Copying sample files to GPU memory space \n");
        hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
        hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
        hipMalloc(&d_conc, sample_size * sizeof(double));
        hipMemcpy(d_STATES_cache, cache, (num_of_states + 2) * sample_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);

        // // Get the maximum number of active blocks per multiprocessor
        // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, do_drug_sim_analytical, threadsPerBlock);

        // // Calculate the total number of blocks
        // int numTotalBlocks = numBlocks * cudaDeviceGetMultiprocessorCount();

        tic();
        printf("Timer started, doing simulation.... \n\n\nGPU Usage at this moment: \n");
        int thread = 32;
        int block = (sample_size + thread - 1) / thread;
        // int block = (sample_size + thread - 1) / thread;
        if (gpu_check(15 * sample_size * sizeof(double) + sizeof(param_t)) == 1) {
            printf("GPU memory insufficient!\n");
            return 0;
        }
        printf("Sample size: %d\n", sample_size);
        hipSetDevice(p_param->gpu_index);
        printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n", block, thread);
        // initscr();
        // printf("[____________________________________________________________________________________________________]  0.00 %% \n");

        kernel_DrugSimulation_postpro<<<block, thread>>>(d_ic50, d_cvar, d_conc, d_CONSTANTS, d_STATES, d_STATES_cache, d_RATES, d_ALGEBRAIC,
                                                 d_mec_CONSTANTS, d_mec_STATES, d_mec_RATES, d_mec_ALGEBRAIC,
                                                 d_STATES_RESULT, d_all_states,
                                                 time, states, dt, cai_result,
                                                 ina, inal,
                                                 ical, ito,
                                                 ikr, iks,
                                                 ik1, tension,
                                                 sample_size,
                                                 temp_result, cipa_result,
                                                 d_p_param);
        // block per grid, threads per block
        // endwin();

        hipDeviceSynchronize();

        printf("allocating memory for computation result in the CPU, malloc style \n");
        double *h_states, *h_time, *h_dt, *h_ical, *h_inal, *h_cai_result, *h_ina, *h_ito, *h_ikr, *h_iks, *h_ik1, *h_tension;
        cipa_t *h_cipa_result;

        h_states = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for STATES, \n");
        h_time = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for time, \n");
        h_dt = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for dt, \n");
        h_cai_result = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for Cai, \n");
        h_ina = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for iNa, \n");
        h_ito = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ito, \n");
        h_ikr = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ikr, \n");
        h_iks = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for iks, \n");
        h_ik1 = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ik1, \n");
        h_ical = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ICaL, \n");
        h_inal = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        h_tension = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        h_cipa_result = (cipa_t *)malloc(sample_size * sizeof(cipa_t));
        printf("...allocating for INaL and postprocessing, all set!\n");

        ////// copy the data back to CPU, and write them into file ////////
        printf("copying the data back to the CPU \n");

        hipMemcpy(h_states, states, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_time, time, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_dt, dt, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ical, ical, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_inal, inal, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_cai_result, cai_result, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ina, ina, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ito, ito, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ikr, ikr, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_iks, iks, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ik1, ik1, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_tension, tension, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_cipa_result, cipa_result, sample_size * sizeof(cipa_t), hipMemcpyDeviceToHost);

        hipFree(d_ALGEBRAIC);
        hipFree(d_CONSTANTS);
        hipFree(d_RATES);
        hipFree(d_STATES);
        hipFree(d_mec_ALGEBRAIC);
        hipFree(d_mec_CONSTANTS);
        hipFree(d_mec_RATES);
        hipFree(d_mec_STATES);
        hipFree(d_p_param);
        hipFree(temp_result);
        hipFree(cipa_result);
        hipFree(d_STATES_RESULT);
        hipFree(d_ic50);
        hipFree(d_cvar);
        hipFree(d_conc);
        hipFree(time);
        hipFree(dt);
        hipFree(states);
        hipFree(ical);
        hipFree(inal);
        hipFree(cai_result);
        hipFree(ina);
        hipFree(ito);
        hipFree(ikr);
        hipFree(iks);
        hipFree(ik1);
        hipFree(tension);
    
        FILE *writer;
        int check;
        bool folder_created = false;

        printf("writing to file... \n");
        // sample loop
        for (int sample_id = 0; sample_id < sample_size; sample_id++) {
            // printf("writing sample %d... \n",sample_id);
            char sample_str[ENOUGH];
            char conc_str[ENOUGH];
            char filename[500] = "./result/post_";
            sprintf(sample_str, "%d", sample_id);
            //sprintf(conc_str, "%.2f", conc[sample_id]);
            strcat(filename, match[1].str().c_str());
            strcat(filename, "/");
            if (folder_created == false) {
                check = mkdir(filename, 0777);
                // check if directory is created or not
                if (!check) {
                    printf("Directory created\n");
                } else {
                    printf("Unable to create directory, or the folder is already created, relax mate...\n");
                }
                folder_created = true;
            }

            strcat(filename, sample_str);
            strcat(filename, "_pace.csv");

            writer = fopen(filename, "w");
            fprintf(writer, "Time,Vm,dVm/dt,Cai,INa,INaL,ICaL,IKs,IKr,IK1,Ito,Tension\n");
            for (int datapoint = 1; datapoint < datapoint_size; datapoint++) {
                if (h_time[ sample_id + (datapoint * sample_size)] == 0.0) {break;}
                fprintf(writer, "%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", // change this into string, or limit the decimal accuracy, so we can decrease filesize
                        h_time[sample_id + (datapoint * sample_size)],
                        h_states[sample_id + (datapoint * sample_size)],
                        h_dt[sample_id + (datapoint * sample_size)],
                        h_cai_result[sample_id + (datapoint * sample_size)],

                        h_ina[sample_id + (datapoint * sample_size)],
                        h_inal[sample_id + (datapoint * sample_size)],

                        h_ical[sample_id + (datapoint * sample_size)],
                        h_iks[sample_id + (datapoint * sample_size)],

                        h_ikr[sample_id + (datapoint * sample_size)],
                        h_ik1[sample_id + (datapoint * sample_size)],

                        h_ito[sample_id + (datapoint * sample_size)],
                        h_tension[sample_id + (datapoint * sample_size)]);
            }
            fclose(writer);
        }

        printf("writing each biomarkers value... \n");
        // sample loop
        // char conc_str[ENOUGH];
        char filename[500] = "./result/post_";
        // sprintf(sample_str, "%d", sample_id);
        // sprintf(conc_str, "%.2f", conc[sample_id]);
        strcat(filename, match[1].str().c_str());
        strcat(filename, "/");
        // printf("creating %s... \n", filename);
        if (folder_created == false) {
            check = mkdir(filename, 0777);
            // check if directory is created or not
            if (!check) {
                printf("Directory created\n");
            } else {
                printf("Unable to create directory, or the folder is already created, relax mate...\n");
            }
            folder_created = true;
        }

        // strcat(filename,sample_str);
        strcat(filename, "_biomarkers.csv");

        writer = fopen(filename, "a");

        fprintf(writer, "sample,qnet,inal_auc,ical_auc,apd90,apd50,apd_tri,cad90,cad50,cad_tri,dvmdt_repol,vm_peak,vm_valley,vm_dia,ca_peak,ca_valley,ca_dia\n");
        for (int sample_id = 0; sample_id < sample_size; sample_id++) {
            // printf("writing sample %d... \n",sample_id);

            fprintf(writer, "%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", // change this into string, or limit the decimal accuracy, so we can decrease filesize
                    sample_id,
                    h_cipa_result[sample_id].qnet,
                    h_cipa_result[sample_id].inal_auc,
                    h_cipa_result[sample_id].ical_auc,

                    h_cipa_result[sample_id].apd90,
                    h_cipa_result[sample_id].apd50,
                    h_cipa_result[sample_id].apd90 - h_cipa_result[sample_id].apd50,

                    h_cipa_result[sample_id].cad90,
                    h_cipa_result[sample_id].cad50,
                    h_cipa_result[sample_id].cad90 - h_cipa_result[sample_id].cad50,

                    h_cipa_result[sample_id].dvmdt_repol,
                    h_cipa_result[sample_id].vm_peak,
                    h_cipa_result[sample_id].vm_valley,
                    h_cipa_result[sample_id].vm_dia,

                    h_cipa_result[sample_id].ca_peak,
                    h_cipa_result[sample_id].ca_valley,
                    h_cipa_result[sample_id].ca_dia

                    //      temp_result[sample_id].qnet = 0.;
                    // temp_result[sample_id].inal_auc = 0.;
                    // temp_result[sample_id].ical_auc = 0.;

                    // temp_result[sample_id].dvmdt_repol = -999;
                    // temp_result[sample_id].dvmdt_max = -999;
                    // temp_result[sample_id].vm_peak = -999;
                    // temp_result[sample_id].vm_valley = d_STATES[(sample_id * num_of_states) +V];
                    // temp_result[sample_id].vm_dia = -999;

                    // temp_result[sample_id].apd90 = 0.;
                    // temp_result[sample_id].apd50 = 0.;
                    // temp_result[sample_id].ca_peak = -999;
                    // temp_result[sample_id].ca_valley = d_STATES[(sample_id * num_of_states) +cai];
                    // temp_result[sample_id].ca_dia = -999;
                    // temp_result[sample_id].cad90 = 0.;
                    // temp_result[sample_id].cad50 = 0.;
            );
        }
        fclose(writer);

        toc();

        return 0; // k bye
    }

    printf("Timer started, doing simulation.... \n\n\nGPU Usage at this moment: \n");
    if (gpu_check(15 * sample_size * datapoint_size * sizeof(double) + sizeof(param_t)) == 1) {
        printf("GPU memory insufficient!\n");
        return 1;
    }
    printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n",
           blocksPerGrid, threadsPerBlock);

    kernel_DrugSimulation<<<blocksPerGrid, threadsPerBlock>>>(
        d_ic50, d_cvar, d_conc, d_CONSTANTS, d_STATES, d_STATES_init, d_RATES, d_ALGEBRAIC, d_STATES_RESULT,
        d_mec_CONSTANTS, d_mec_STATES, d_mec_RATES, d_mec_ALGEBRAIC, time, states, dt, cai_result, ina, inal, ical, ito,
        ikr, iks, ik1, sample_size, temp_result, cipa_result, d_p_param);
    hipDeviceSynchronize();

    double *h_states = (double *)malloc(ORd_num_of_states * sample_size * sizeof(double));
    cipa_t *h_cipa_result = (cipa_t *)malloc(sample_size * sizeof(cipa_t));

    printf("copying the data back to the CPU \n");
    hipMemcpy(h_states, d_STATES_RESULT, sample_size * ORd_num_of_states * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_cipa_result, cipa_result, sample_size * sizeof(cipa_t), hipMemcpyDeviceToHost);

    write_results_to_file("./result", drug_name, h_states, h_cipa_result, sample_size, ORd_num_of_states);

    freeingMemory(d_ALGEBRAIC, d_CONSTANTS, d_RATES, d_STATES, d_mec_ALGEBRAIC, d_mec_CONSTANTS, d_mec_RATES,
                  d_mec_STATES, d_p_param, temp_result, cipa_result, d_STATES_RESULT, d_ic50, ic50, conc, h_states,
                  h_cipa_result, p_param);

    toc();

    return 0;
}
